#include "hip/hip_runtime.h"
#ifdef __cplusplus
extern "C++" {
#include "glife.h"
}
#include <hip/hip_runtime.h>

// HINT: YOU CAN USE THIS METHOD FOR ERROR CHECKING
// Print error message on CUDA API or kernel launch
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                    msg, hipGetErrorString(__err), \
                    __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
        } \
    } while (0)

// TODO: YOU MAY NEED TO USE IT OR CREATE MORE
__device__ int getNeighbors(int* grid, int tot_rows, int tot_cols,
        int row, int col) {
    return numOfNeighbors;
}

// TODO: YOU NEED TO IMPLEMENT KERNEL TO RUN ON GPU DEVICE 
__global__ void kernel()
{

}

// TODO: YOU NEED TO IMPLEMENT TO PRINT THE INDEX RESULTS 
void cuda_dump()
{
    printf("===============================\n");

    printf("===============================\n");
}

// TODO: YOU NEED TO IMPLEMENT TO PRINT THE INDEX RESULTS 
void cuda_dump_index()
{
    printf(":: Dump Row Column indices\n");
}

// TODO: YOU NEED TO IMPLEMENT ON CUDA VERSION
uint64_t runCUDA(int rows, int cols, int gen, 
                 GameOfLifeGrid* g_GameOfLifeGrid, int display);
{
    hipSetDevice(0); // DO NOT CHANGE THIS LINE 

    uint64_t difft;

    // ---------- TODO: CALL CUDA API HERE ----------


    // Start timer for CUDA kernel execution
    difft = dtime_usec(0);
    // ----------  TODO: CALL KERNEL HERE  ----------


    // Finish timer for CUDA kernel execution
    difft = dtime_usec(difft);

    // Print the results
    if (display) {
        cuda_dump();
        cuda_dump_index();
    }
    return difft;
}
#endif
